
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <utility>

void cuda_err_check (hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf (stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString (err), file, line);
        exit (EXIT_FAILURE);
    }
}

__global__ void init (int *d_a, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;
    d_a[idx] = idx;
}

void init_array(int *d_a, int n){
    int lws = 256;
    int nblks = (n + lws - 1) / lws;
    init<<<nblks, lws>>>(d_a, n);
}

__global__ void smooth_v4 (int4 * d_in, int4 * d_out, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int4 sum = d_in[idx];
    int4 prev = {0, sum.x, sum.y, sum.z};
    int4 next = {sum.y, sum.z, sum.w, 0};
    int4 div = make_int4(2, 3, 3, 2);
    if (idx > 0  ) {prev.x = d_in[idx-1].w; div.x++;}
    if (idx < n-1) {next.w = d_in[idx+1].x; div.w++;}
    d_out[idx] = {(sum.x + prev.x + next.x) / div.x,
                  (sum.y + prev.y + next.y) / div.y,
                  (sum.z + prev.z + next.z) / div.z,
                  (sum.w + prev.w + next.w) / div.w};
}

// DOMANDA: cuda non supporta int8 o int16?
// __global__ void smooth_v8(int8 *d_in, int8 *d_out, int n){
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     int8 sum = d_in[idx];
//     int8 prev = {0, sum.x, sum.y, sum.z, sum.w, 0, 0, 0};
//     int8 next = {sum.y, sum.z, sum.w, sum.x, sum.y, sum.z, sum.w, 0};
//     int8 div = make_int8(2, 3, 3, 3, 3, 2, 2, 2);
//     if (idx > 0  ) {prev.x = d_in[idx-1].w; div.x++;}
//     if (idx < n-1) {next.w = d_in[idx+1].x; div.w++;}
//     d_out[idx] = {(sum.x + prev.x + next.x) / div.x,
//                   (sum.y + prev.y + next.y) / div.y,
//                   (sum.z + prev.z + next.z) / div.z,
//                   (sum.w + prev.w + next.w) / div.w,
//                   (sum.x + prev.x + next.x) / div.x,
//                   (sum.y + prev.y + next.y) / div.y,
//                   (sum.z + prev.z + next.z) / div.z,
//                   (sum.w + prev.w + next.w) / div.w};
// }

void smooth_array_v4(int * d_in, int * d_out, int n){
    n /= 4;
    int lws = 256;
    int nblks = (n + lws - 1) / lws;
    smooth_v4<<<nblks, lws>>>((int4*)d_in, (int4*)d_out, n);
}

void verify (int *h_a, int nels){
    for (int i = 0; i < nels; ++i) {
        int expected = i - !!(i == nels - 1);

        if (h_a[i] != expected) {
            fprintf(stderr, "mismatch @ %d: %d != %d\n",
            i, h_a[i], expected);
        }
    }
}

int main (int argc, char * argv[]){
    if (argc != 2){
        printf("Usage: %s <nels>", argv[0]);
    }
    int nels = atoi(argv[1]);
    int *h_a, *d_in, *d_out;
    float t1;
    hipError_t err;

    size_t memsize = nels*sizeof(int);

    hipEvent_t pre_init, post_init, post_smooth, post_memcpy, post_verify;
    err = hipEventCreate(&pre_init);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_init);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_smooth);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_memcpy);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_verify);
    cuda_err_check(err, __FILE__, __LINE__);

    //  finished with event creation
    
    err = hipMalloc    (&d_in, memsize);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMalloc    (&d_out, memsize);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipHostMalloc(&h_a, memsize, hipHostMallocDefault);
    cuda_err_check(err, __FILE__, __LINE__);

    // finished with memory allocation

    err = hipEventRecord(pre_init);
    cuda_err_check(err, __FILE__, __LINE__);

    init_array(d_in, nels);

    err = hipEventRecord(post_init);
    cuda_err_check(err, __FILE__, __LINE__);

    smooth_array_v4(d_in, d_out, nels);

    err = hipEventRecord(post_smooth);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(h_a, d_out, memsize, hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventRecord(post_memcpy);
    cuda_err_check(err, __FILE__, __LINE__);

    verify(h_a, nels);

    err = hipEventRecord(post_verify);
    cuda_err_check(err, __FILE__, __LINE__);

    // DOMANDA: perché nella veraione OpenCL moltiplichiamo memsize per 2?
    hipEventElapsedTime(&t1, pre_init, post_init);
    printf("init : %f ms (%f GB/s)\n", t1, memsize/t1/1e6);

    // DOMANDA: perché moltiplichiamo per 1.5 e abbiamo un uso della bandwidth terribile?
    hipEventElapsedTime(&t1, post_init, post_smooth);
    printf("smooth : %f ms (%f GB/s)\n", t1, 1.5*memsize/t1/1e6);

    hipEventElapsedTime(&t1, post_smooth, post_memcpy);
    printf("memcp: %f ms (%f GB/s)\n", t1, memsize/t1/1e6);

    hipEventElapsedTime(&t1, post_memcpy, post_verify);
    printf("check: %f ms\n", t1);

    hipFree(d_in);
    hipFree(d_out);
    hipHostFree(h_a);
    hipEventDestroy(pre_init);
    hipEventDestroy(post_init);
    hipEventDestroy(post_smooth);
    hipEventDestroy(post_memcpy);
    hipEventDestroy(post_verify);
}