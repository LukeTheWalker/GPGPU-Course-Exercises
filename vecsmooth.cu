
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <utility>

void cuda_err_check (hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf (stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString (err), file, line);
        exit (EXIT_FAILURE);
    }
}

__global__ void init (int *d_a, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;
    d_a[idx] = idx;
}

void init_array(int *d_a, int n){
    int lws = 256;
    int nblks = (n + lws - 1) / lws;
    init<<<nblks, lws>>>(d_a, n);
}

// DOMANDA: con restrict non ho assolutamente alcun guadagno di performance
__global__ void smooth (int * d_in, int * d_out, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int div = 1;
    int sum = d_in[idx];
    if (idx > 0) {sum += d_in[idx-1]; div++;}
    if (idx < n-1) {sum += d_in[idx+1]; div++;}
    d_out[idx] = sum / div;
}

void smooth_array(int * d_in, int * d_out, int n){
    int lws = 256;
    int nblks = (n + lws - 1) / lws;
    smooth<<<nblks, lws>>>(d_in, d_out, n);
}

void verify (int *h_a, int nels){
    for (int i = 0; i < nels; ++i) {
        int expected = i - !!(i == nels - 1);

        if (h_a[i] != expected) {
            fprintf(stderr, "mismatch @ %d: %d != %d\n",
            i, h_a[i], expected);
        }
    }
}

int main (int argc, char * argv[]){
    if (argc != 2){
        printf("Usage: %s <nels>", argv[0]);
    }
    int nels = atoi(argv[1]);
    int *h_a, *d_in, *d_out;
    float t1;
    hipError_t err;

    size_t memsize = nels*sizeof(int);

    hipEvent_t pre_init, post_init, post_smooth, post_memcpy, post_verify;
    err = hipEventCreate(&pre_init);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_init);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_smooth);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_memcpy);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_verify);
    cuda_err_check(err, __FILE__, __LINE__);

    //  finished with event creation
    
    err = hipMalloc    (&d_in, memsize);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMalloc    (&d_out, memsize);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipHostMalloc(&h_a, memsize, hipHostMallocDefault);
    cuda_err_check(err, __FILE__, __LINE__);

    // finished with memory allocation

    err = hipEventRecord(pre_init);
    cuda_err_check(err, __FILE__, __LINE__);

    init_array(d_in, nels);

    err = hipEventRecord(post_init);
    cuda_err_check(err, __FILE__, __LINE__);

    smooth_array(d_in, d_out, nels);

    err = hipEventRecord(post_smooth);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(h_a, d_out, memsize, hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventRecord(post_memcpy);
    cuda_err_check(err, __FILE__, __LINE__);

    verify(h_a, nels);

    err = hipEventRecord(post_verify);
    cuda_err_check(err, __FILE__, __LINE__);

    hipEventElapsedTime(&t1, pre_init, post_init);
    printf("init : %f ms (%f GB/s)\n", t1, memsize/t1/1e6);

    hipEventElapsedTime(&t1, post_init, post_smooth);
    printf("smooth : %f ms (%f GB/s)\n", t1, 4*memsize/t1/1e6);

    hipEventElapsedTime(&t1, post_smooth, post_memcpy);
    printf("memcp: %f ms (%f GB/s)\n", t1, memsize/t1/1e6);

    hipEventElapsedTime(&t1, post_memcpy, post_verify);
    printf("check: %f ms\n", t1);

    hipFree(d_in);
    hipFree(d_out);
    hipHostFree(h_a);
    hipEventDestroy(pre_init);
    hipEventDestroy(post_init);
    hipEventDestroy(post_smooth);
    hipEventDestroy(post_memcpy);
    hipEventDestroy(post_verify);
}