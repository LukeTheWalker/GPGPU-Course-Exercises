
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <utility>

__device__ void print_int4(int4 a) {
    printf("%d %d %d %d\n", a.x, a.y, a.z, a.w);
}

void cuda_err_check (hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf (stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString (err), file, line);
        exit (EXIT_FAILURE);
    }
}

__global__ void init (int *d_a, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;
    d_a[idx] = idx;
}

void init_array(int *d_a, int n){
    int lws = 256;
    int nblks = (n + lws - 1) / lws;
    init<<<nblks, lws>>>(d_a, n);
}


__global__ void smooth_lmem_v4 (int4 * d_in, int4 * d_out, int n){
    extern __shared__ int lmem[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;
    int * lprev = lmem;
    int * lnext = lmem + blockDim.x;
    int lidx = threadIdx.x;

    int4 div = {2,3,3,2};

    int4 sum = d_in[idx];

    if (lidx != blockDim.x - 1)
        lprev[lidx + 1] = sum.w;

    if (lidx != 0)
        lnext[lidx - 1] = sum.x;

    __syncthreads();

    int4 prev = {0, sum.x, sum.y, sum.z};
    int4 next = {sum.y, sum.z, sum.w, 0};

    if (idx > 0) {
        if (lidx != 0) 
            prev.x = lprev[lidx];
        else 
            prev.x = d_in[idx-1].w; 
        div.x++;
    }
    if (idx < n-1) {
        if (lidx != blockDim.x - 1) 
            next.w = lnext[lidx];
        else 
            next.w = d_in[idx+1].x; 
        div.w++;
    }
    
    d_out[idx] = {(sum.x + prev.x + next.x) / div.x,
                  (sum.y + prev.y + next.y) / div.y,
                  (sum.z + prev.z + next.z) / div.z,
                  (sum.w + prev.w + next.w) / div.w};
}

__global__ void smooth_lmem_oversize_v4 (int4 * d_in, int4 * d_out, int n){
    extern __shared__ int lmem_oversize[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;
    int lidx = threadIdx.x;
    int * lprev = lmem_oversize;
    int * lnext = lmem_oversize + blockDim.x;

    int4 div = {2,3,3,2};

    int4 sum = d_in[idx];


    if (lidx != blockDim.x - 1)
        lprev[lidx + 1] = sum.w;

    if (lidx != 0)
        lnext[lidx - 1] = sum.x;

    if (idx != 0 && lidx == 0)
        lprev[lidx] = d_in[idx - 1].w;        
    
    if (idx != n - 1 && lidx == blockDim.x - 1)
        lnext[lidx] = d_in[idx + 1].x;        

    __syncthreads();

    int4 prev = {0, sum.x, sum.y, sum.z};
    int4 next = {sum.y, sum.z, sum.w, 0};

    if (idx != 0) {
        prev.x = lprev[lidx]; 
        div.x++;
    }
    if (idx != n - 1) {
        next.w = lnext[lidx]; 
        div.w++;
    }

    d_out[idx] = {(sum.x + prev.x + next.x) / div.x,
                  (sum.y + prev.y + next.y) / div.y,
                  (sum.z + prev.z + next.z) / div.z,
                  (sum.w + prev.w + next.w) / div.w};
}

void smooth_array(int * d_in, int * d_out, int n){
    int lws = 256;
    n /= 4;
    int nblks = (n + lws - 1) / lws;
#if 1
    // printf("shared memory size = %lu\n", lws * sizeof(int4) + 2 * sizeof(int));
    smooth_lmem_oversize_v4<<<nblks, lws, 2 + lws * sizeof(int)>>>((int4 *)d_in, (int4 *)d_out, n);
#else
    smooth_lmem_v4<<<nblks, lws, lws * sizeof(int)>>>((int4 *)d_in, (int4 *)d_out, n);
#endif
}

void verify (int *h_a, int nels){
    for (int i = 0; i < nels; ++i) {
        int expected = i - !!(i == nels - 1);

        if (h_a[i] != expected) {
            fprintf(stderr, "mismatch @ %d: %d != %d\n",
            i, h_a[i], expected);
        }
    }
}

int main (int argc, char * argv[]){
    if (argc != 2){
        printf("Usage: %s <nels>", argv[0]);
    }
    int nels = atoi(argv[1]);
    int *h_a, *d_in, *d_out;
    float t1;
    hipError_t err;

    size_t memsize = nels*sizeof(int);

    hipEvent_t pre_init, post_init, post_smooth, post_memcpy, post_verify;
    err = hipEventCreate(&pre_init);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_init);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_smooth);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_memcpy);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventCreate(&post_verify);
    cuda_err_check(err, __FILE__, __LINE__);

    //  finished with event creation
    
    err = hipMalloc    (&d_in, memsize);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMalloc    (&d_out, memsize);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipHostMalloc(&h_a, memsize, hipHostMallocDefault);
    cuda_err_check(err, __FILE__, __LINE__);

    // finished with memory allocation

    err = hipEventRecord(pre_init);
    cuda_err_check(err, __FILE__, __LINE__);

    init_array(d_in, nels);

    err = hipEventRecord(post_init);
    cuda_err_check(err, __FILE__, __LINE__);

    smooth_array(d_in, d_out, nels);

    err = hipEventRecord(post_smooth);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(h_a, d_out, memsize, hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventRecord(post_memcpy);
    cuda_err_check(err, __FILE__, __LINE__);

    verify(h_a, nels);

    err = hipEventRecord(post_verify);
    cuda_err_check(err, __FILE__, __LINE__);

    hipEventElapsedTime(&t1, pre_init, post_init);
    printf("init : %f ms (%f GB/s)\n", t1, memsize/t1/1e6);

    hipEventElapsedTime(&t1, post_init, post_smooth);
    printf("smooth : %f ms (%f GB/s)\n", t1, 4*memsize/t1/1e6);

    hipEventElapsedTime(&t1, post_smooth, post_memcpy);
    printf("memcp: %f ms (%f GB/s)\n", t1, memsize/t1/1e6);

    hipEventElapsedTime(&t1, post_memcpy, post_verify);
    printf("check: %f ms\n", t1);

    hipFree(d_in);
    hipFree(d_out);
    hipHostFree(h_a);
    hipEventDestroy(pre_init);
    hipEventDestroy(post_init);
    hipEventDestroy(post_smooth);
    hipEventDestroy(post_memcpy);
    hipEventDestroy(post_verify);
}